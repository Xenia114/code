#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void bitonicSortStep(int* data, int j, int k, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    int ixj = i ^ j;
    if (ixj > i) {
        if (((i & k) == 0 && data[i] > data[ixj]) ||
            ((i & k) != 0 && data[i] < data[ixj])) {
            int temp = data[i];
            data[i] = data[ixj];
            data[ixj] = temp;
        }
    }
}

void bitonicSort(int* data, int size, int threads) {
    int* dev_data;
    hipMalloc((void**)&dev_data, size * sizeof(int));
    hipMemcpy(dev_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (size + threads - 1) / threads;

    for (int k = 2; k <= size; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            bitonicSortStep<<<blocks, threads>>>(dev_data, j, k, size);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_data);
}

int main(int argc, char** argv) {
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <ARRAY_SIZE> <THREADS_PER_BLOCK>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);
    int threads = atoi(argv[2]);

    // Проверка: размер должен быть степенью двойки
    if ((size & (size - 1)) != 0) {
        fprintf(stderr, "Error: size must be a power of 2\n");
        return 1;
    }

    int* data = (int*)malloc(size * sizeof(int));
    if (!data) {
        fprintf(stderr, "Memory allocation failed\n");
        return 1;
    }

    srand((unsigned)time(NULL));
    for (int i = 0; i < size; ++i) {
        data[i] = rand() % 10000;
    }

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    bitonicSort(data, size, threads);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%.6f\n", milliseconds / 1000.0f);

    free(data);
    return 0;
}
