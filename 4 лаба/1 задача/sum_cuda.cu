#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define ARRAY_SIZE 10000000
#define THREADS_PER_BLOCK 256

__global__ void sum_reduction(int* input, long long* partial_sums, int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? input[i] : 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        partial_sums[blockIdx.x] = sdata[0];
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <num_runs> <threads_per_block>\n", argv[0]);
        return 1;
    }

    int runs = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);

    int* h_array = (int*)malloc(ARRAY_SIZE * sizeof(int));
    if (!h_array) {
        fprintf(stderr, "Host memory allocation failed!\n");
        return 1;
    }

    for (int i = 0; i < ARRAY_SIZE; ++i) {
        h_array[i] = i + 1;
    }

    int* d_array;
    long long* d_partial_sums;
    hipMalloc((void**)&d_array, ARRAY_SIZE * sizeof(int));
    hipMemcpy(d_array, h_array, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (ARRAY_SIZE + threads_per_block - 1) / threads_per_block;
    hipMalloc((void**)&d_partial_sums, blocks * sizeof(long long));

    long long total_time = 0;

    for (int run = 0; run < runs; ++run) {
        hipDeviceSynchronize();
        clock_t start = clock();

        sum_reduction<<<blocks, threads_per_block, threads_per_block * sizeof(int)>>>(d_array, d_partial_sums, ARRAY_SIZE);
        hipDeviceSynchronize();

        long long* h_partial_sums = (long long*)malloc(blocks * sizeof(long long));
        hipMemcpy(h_partial_sums, d_partial_sums, blocks * sizeof(long long), hipMemcpyDeviceToHost);

        long long total_sum = 0;
        for (int i = 0; i < blocks; ++i)
            total_sum += h_partial_sums[i];

        clock_t end = clock();
        total_time += (end - start);

        free(h_partial_sums);
    }

    double avg_time = (double)total_time / runs / CLOCKS_PER_SEC;
    printf("%.6f\n", avg_time);  // Для вывода в скрипт

    free(h_array);
    hipFree(d_array);
    hipFree(d_partial_sums);

    return 0;
}
